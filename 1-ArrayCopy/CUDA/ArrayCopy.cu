#include "hip/hip_runtime.h"
/*
 * cudaTutorial
 * ArrayCopy.cu
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Copyright (c) 2021 Hendrik Schwanekamp
 *
 */

#include <iostream>
#include "../../shared/dataGenerator.h"
#include "../../shared/cudaErrorCheck.h"

constexpr int numElements = 1e8;

void __global__ copyArray(const float* in, float* out, int num)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for(int i=id; i < num; i+=stride) {
        out[i] = in[i];
    }
}

int main () {
    float* input;
    float* output;
    gpuErrchk( hipMallocManaged(&input, numElements*sizeof(float)));
    gpuErrchk( hipMallocManaged(&output, numElements*sizeof(float)));

    genRandomData(input, input+numElements);

    copyArray<<<4096,256>>>(input, output, numElements);
    gpuErrchk( hipPeekAtLastError());
    gpuErrchk( hipDeviceSynchronize());

    for(int i=0; i<10; i++)
        std::cout << output[i] << std::endl;

    hipFree(input);
    hipFree(output);
    return 0;
}