/*
 * cudaTutorial
 * SharedMemory.cu
 *
 * @author: Hendrik Schwanekamp
 * @mail:   hendrik.schwanekamp@gmx.net
 *
 * Copyright (c) 2021 Hendrik Schwanekamp
 *
 */

#include <iostream>
#include <hipcub/hipcub.hpp>
#include "../shared/dataGenerator.h"
#include "../shared/cudaErrorCheck.h"

constexpr int numElements = 1e8;

int main () {
    float* input;
    float* output;
    gpuErrchk( hipMallocManaged(&input, numElements*sizeof(float)));
    gpuErrchk( hipMallocManaged(&output, numElements*sizeof(float)));

    genRandomData(input, input+numElements);

    // Determine temporary device storage requirements
    // this is done by passing 0 as the temp storage
    void     *tempStorage_d = NULL;
    size_t   tempStorageSize = 0;
    gpuErrchk( hipcub::DeviceRadixSort::SortKeys(tempStorage_d, tempStorageSize, input, output, numElements));

    // Allocate temporary storage
    gpuErrchk( hipMalloc(&tempStorage_d, tempStorageSize));

    // Run sorting operation
    gpuErrchk( hipcub::DeviceRadixSort::SortKeys(tempStorage_d, tempStorageSize, input, output, numElements));

    gpuErrchk( hipDeviceSynchronize());
    for(int i=0; i<10; i++)
        std::cout << output[i] << std::endl;

    hipFree(input);
    hipFree(output);
    return 0;
}